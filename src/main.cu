#include "hip/hip_runtime.h"
// Based on Imgui example for GLFW + OpenGL3
// Read online: https://github.com/ocornut/imgui/tree/master/docs

#define GL_SILENCE_DEPRECATION
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>

#include <glad/glad.h>

#include <GLFW/glfw3.h> // Will drag system OpenGL headers

#include <glm/vec3.hpp>
#include <glm/geometric.hpp>

#include <iostream>
#include <vector>

#include "shader.h"
#include "buffer.h"


#define GRAVITY 6.67408e-6f
#define SOFTENING 1e-3f

#define BLOCK_SIZE 512

static void glfw_error_callback(int error, const char* description)
{
	std::cerr << "GLFW Error " << error << ": " << description << std::endl;
}

static void simulate(std::vector<Point>& points, float dt)
{
	using namespace glm;

	for (auto& p1 : points)
	{
		auto acceleration = vec3(0.0f);
		for (auto& p2 : points)
		{
			if (&p1 == &p2)
				continue;

			auto r = p2.position - p1.position;
			auto distSqr = dot(r, r);
			auto force = GRAVITY * p2.mass / (distSqr + SOFTENING);


			acceleration += force * normalize(r);
		}

		p1.velocity += acceleration * dt;
	}


}

static void update(std::vector<Point>& points, float dt)
{
	for (auto& p : points)
	{
		p.position += p.velocity * dt;
	}
}

__global__ void simulate_cuda(float* points, float dt, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;
	float p_x = points[idx * 7];
	float p_y = points[idx * 7 + 1];
	float p_z = points[idx * 7 + 2];


	float a_x = 0.0f;
	float a_y = 0.0f;
	float a_z = 0.0f;

	for (int i = 0; i < n; i++)
	{
		if (i == idx)
			continue;


		float r[3] = { points[i * 7] - p_x, points[i * 7 + 1] - p_y, points[i * 7 + 2] - p_z };
		float distSqr = r[0] * r[0] + r[1] * r[1] + r[2] * r[2];
		float dist = sqrtf(distSqr);
		float force = GRAVITY * points[7*i + 6] / (distSqr + SOFTENING);

		float r_unit[3] = { r[0] / dist, r[1] / dist, r[2] / dist };

		a_x += force * r_unit[0];
		a_y += force * r_unit[1];
		a_z += force * r_unit[2];
	}

	points[idx*7 + 3] += a_x * dt;
	points[idx*7 + 4] += a_y * dt;
	points[idx*7 + 5] += a_z * dt;

}

__global__ void update_cuda(float* points, float dt, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;
	points[idx * 7] += points[idx * 7 + 3] * dt;
	points[idx * 7 + 1] += points[idx * 7 + 4] * dt;
	points[idx * 7 + 2] += points[idx * 7 + 5] * dt;
}

// Main code
int main(int, char**)
{
	glfwSetErrorCallback(glfw_error_callback);
	if (!glfwInit())
		return 1;

	// Decide GL+GLSL versions
#if defined(IMGUI_IMPL_OPENGL_ES2)
	// GL ES 2.0 + GLSL 100
	const char* glsl_version = "#version 100";
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
	glfwWindowHint(GLFW_CLIENT_API, GLFW_OPENGL_ES_API);
#elif defined(__APPLE__)
	// GL 3.2 + GLSL 150
	const char* glsl_version = "#version 330";
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // Required on Mac
#else
	// GL 3.0 + GLSL 130
	const char* glsl_version = "#version 130";
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
	//glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
	//glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // 3.0+ only
#endif

	glfwWindowHint(GLFW_RESIZABLE, GLFW_FALSE);
	// Create window with graphics context
	GLFWwindow* window = glfwCreateWindow(800, 800, "Tarea 3", nullptr, nullptr);
	if (window == nullptr)
		return 1;
	glfwMakeContextCurrent(window);
	glfwSwapInterval(1); // Enable vsync

	// glad: load all OpenGL function pointers
	// ---------------------------------------
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		return -1;
	}

	// Setup Dear ImGui context
	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO();
	(void)io;
	io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
	io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;      // Enable Gamepad Controls

	// Setup Dear ImGui style
	ImGui::StyleColorsDark();

	// Setup Platform/Renderer backends
	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init(glsl_version);

	// Our state
	bool use_quads = false;
	bool only_points = true;
	bool pause = true;
	bool use_gpu = false;

	float mass = 30.0f;
	float ratio = 0.1f;
	int n = 100;
	ImVec4 clear_color = ImVec4(0.0f, 0.0f, 0.0f, 1.00f);

	// Create a shader program
	Shader shader("shaders/nbody.vert", "shaders/nbody.geom", "shaders/nbody.frag");
	Shader shader_points("shaders/pnbody.vert", "shaders/nbody.frag");
	Buffer buffer;
	buffer.setLayout(shader);
	buffer.setLayout(shader_points);

	auto now = static_cast<float>(glfwGetTime());
	auto last = now;
	float delta;

	while (!glfwWindowShouldClose(window))
	{
		// simulation
		now = static_cast<float>(glfwGetTime());
		delta = now - last;
		last = now;


		if (!pause)
		{
			if (use_gpu)
			{
				buffer.map();
				simulate_cuda << < (n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >> > (buffer.getCudaPoints(), delta, n);
				update_cuda << < (n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >> > (buffer.getCudaPoints(), delta, n);
				buffer.unmap();
			}
			else
			{
				simulate(buffer.data, delta);
				update(buffer.data, delta);
			}

		}



		glfwPollEvents();
		// Start the Dear ImGui frame
		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

		// 2. Show a simple window that we create ourselves. We use a Begin/End pair to create a named window.
		{
			ImGui::Begin("Simulation options");                          // Create a window called "Hello, world!" and append into it.

//			ImGui::Checkbox("Pause", &pause);      // Edit bools storing our window open/close state

			ImGui::Checkbox("Only points?", &only_points);      // Edit bools storing our window open/close state
			ImGui::SameLine();
			ImGui::Checkbox("View quads?", &use_quads);      // Edit bools storing our window open/close state
			ImGui::SameLine();
			ImGui::Checkbox("Use GPU?", &use_gpu);      // Edit bools storing our window open/close state

			ImGui::SliderInt("Number of points", &n, 0, 5000);
			// Vec2 input
			ImGui::SliderFloat("Special mass", &mass, 1.0f, 500.0f, "%.3f");

			// ImGui::ColorEdit("Special Gravity", &acceleration, 0.1f, 1.0f, "%.3f");
			ImGui::SliderFloat("SM Ratio",
				&ratio,
				0.0f,
				1.0f);            // Edit 1 float using a slider from 0.0f to 1.0f
			ImGui::ColorEdit3("Clear color", (float*)&clear_color); // Edit 3 floats representing a color

			const std::string& label = pause ? "Play" : "Pause";
			if (ImGui::Button(label.c_str()))
				pause = !pause;

			ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / io.Framerate, io.Framerate);
			ImGui::End();
		}


		buffer.setSize(n);
		buffer.setSpecialMass(mass);
		buffer.setRatio(ratio);
		buffer.build();

		// Rendering
		ImGui::Render();
		glClear(GL_COLOR_BUFFER_BIT);
		glClearColor(clear_color.x * clear_color.w,
			clear_color.y * clear_color.w,
			clear_color.z * clear_color.w,
			clear_color.w);

		if (only_points)
		{
			shader_points.bind();
			buffer.bind();

			glDrawElements(GL_POINTS, n, GL_UNSIGNED_INT, 0);

			buffer.unbind();
			shader_points.unbind();
		}
		else
		{
			shader.bind();
			shader.setUniform1i("useQuads", use_quads);
			buffer.bind();

			glDrawElements(GL_POINTS, n, GL_UNSIGNED_INT, 0);

			buffer.unbind();
			shader.unbind();
		}



		int display_w, display_h;
		glfwGetFramebufferSize(window, &display_w, &display_h);
		glViewport(0, 0, display_w, display_h);

		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

		glfwSwapBuffers(window);
	}

	// Cleanup
	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();

	glfwDestroyWindow(window);
	glfwTerminate();

	return 0;
}